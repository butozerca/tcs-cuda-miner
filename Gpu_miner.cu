
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring>
#include <fstream>
#include <string>

#define BASE_OFFSET 256
#define THREAD_SIZE 564

#define M_BLOCK_OFFSET 0
#define M_H_OFFSET 128
#define W_OFFSET 160
#define WV_OFFSET 416
#define NONCE_INPUT_OFFSET 448
#define DIGEST_OFFSET 528

#define THREAD_VAR(offset) (shared_mem + BASE_OFFSET + THREAD_SIZE * threadIdx.x + (offset))

#define M_BLOCK THREAD_VAR(M_BLOCK_OFFSET)
#define M_H ((unsigned int *)(THREAD_VAR(M_H_OFFSET)))
#define W ((unsigned int *)(THREAD_VAR(W_OFFSET)))
#define WV ((unsigned int *)(THREAD_VAR(WV_OFFSET)))
#define NONCE_INPUT ((unsigned char*)(THREAD_VAR(NONCE_INPUT_OFFSET)))
#define DIGEST THREAD_VAR(DIGEST_OFFSET)

 
class SHA256
{
protected:
    typedef unsigned char uint8;
    typedef unsigned int uint32;
    typedef unsigned long long uint64;
 
public:
    __device__
    void init(char* shared_mem);
    __device__
    void update(const unsigned char *message, unsigned int len, char* shared_mem);
    __device__
    void final(unsigned char *digest, char* shared_mem);
 
protected:
    __device__
    void transform(const unsigned char *message, unsigned int block_nb, char* shared_mem);
    unsigned int m_tot_len;
    unsigned int m_len;
};

__device__
void sha256(const char* s, int length, char* out);
 
#define SHA2_SHFR(x, n)    (x >> n)
#define SHA2_ROTR(x, n)   ((x >> n) | (x << ((sizeof(x) << 3) - n)))
#define SHA2_ROTL(x, n)   ((x << n) | (x >> ((sizeof(x) << 3) - n)))
#define SHA2_CH(x, y, z)  ((x & y) ^ (~x & z))
#define SHA2_MAJ(x, y, z) ((x & y) ^ (x & z) ^ (y & z))
#define SHA256_F1(x) (SHA2_ROTR(x,  2) ^ SHA2_ROTR(x, 13) ^ SHA2_ROTR(x, 22))
#define SHA256_F2(x) (SHA2_ROTR(x,  6) ^ SHA2_ROTR(x, 11) ^ SHA2_ROTR(x, 25))
#define SHA256_F3(x) (SHA2_ROTR(x,  7) ^ SHA2_ROTR(x, 18) ^ SHA2_SHFR(x,  3))
#define SHA256_F4(x) (SHA2_ROTR(x, 17) ^ SHA2_ROTR(x, 19) ^ SHA2_SHFR(x, 10))
#define SHA2_UNPACK32(x, str)                 \
{                                             \
    *((str) + 3) = (uint8) ((x)      );       \
    *((str) + 2) = (uint8) ((x) >>  8);       \
    *((str) + 1) = (uint8) ((x) >> 16);       \
    *((str) + 0) = (uint8) ((x) >> 24);       \
}
#define SHA2_PACK32(str, x)                   \
{                                             \
    *(x) =   ((uint32) *((str) + 3)      )    \
           | ((uint32) *((str) + 2) <<  8)    \
           | ((uint32) *((str) + 1) << 16)    \
           | ((uint32) *((str) + 0) << 24);   \
}

__device__
void SHA256::transform(const unsigned char *message, unsigned int block_nb, char* shared_mem)
{
    uint32* w = W;
    uint32* wv = WV;
    unsigned int* m_h = M_H;
    unsigned int* sha256_k = (unsigned int*)shared_mem;
    uint32 t1, t2;
    const unsigned char *sub_block;
    int i;
    int j;
    for (i = 0; i < (int) block_nb; i++) {
        sub_block = message + (i << 6);
        for (j = 0; j < 16; j++) {
            SHA2_PACK32(&sub_block[j << 2], &w[j]);
        }
        for (j = 16; j < 64; j++) {
            w[j] =  SHA256_F4(w[j -  2]) + w[j -  7] + SHA256_F3(w[j - 15]) + w[j - 16];
        }
        for (j = 0; j < 8; j++) {
            wv[j] = m_h[j];
        }
        for (j = 0; j < 64; j++) {
            t1 = wv[7] + SHA256_F2(wv[4]) + SHA2_CH(wv[4], wv[5], wv[6])
                + sha256_k[j] + w[j];
            t2 = SHA256_F1(wv[0]) + SHA2_MAJ(wv[0], wv[1], wv[2]);
            wv[7] = wv[6];
            wv[6] = wv[5];
            wv[5] = wv[4];
            wv[4] = wv[3] + t1;
            wv[3] = wv[2];
            wv[2] = wv[1];
            wv[1] = wv[0];
            wv[0] = t1 + t2;
        }
        for (j = 0; j < 8; j++) {
            m_h[j] += wv[j];
        }
    }
}

__device__
void SHA256::init(char* shared_mem)
{
    unsigned int *m_h = M_H;
    m_h[0] = 0x6a09e667;
    m_h[1] = 0xbb67ae85;
    m_h[2] = 0x3c6ef372;
    m_h[3] = 0xa54ff53a;
    m_h[4] = 0x510e527f;
    m_h[5] = 0x9b05688c;
    m_h[6] = 0x1f83d9ab;
    m_h[7] = 0x5be0cd19;
    m_len = 0;
    m_tot_len = 0;
}

__device__
void SHA256::update(const unsigned char *message, unsigned int len, char* shared_mem)
{
    unsigned int block_nb;
    unsigned int new_len, rem_len, tmp_len;
    const unsigned char *shifted_message;
    char* m_block = M_BLOCK;
    tmp_len = 64 - m_len;
    rem_len = len < tmp_len ? len : tmp_len;
    memcpy(&m_block[m_len], message, rem_len);
    if (m_len + len < 64) {
        m_len += len;
        return;
    }
    new_len = len - rem_len;
    block_nb = new_len / 64;
    shifted_message = message + rem_len;
    transform((unsigned char*)m_block, 1, shared_mem);
    transform(shifted_message, block_nb, shared_mem);
    rem_len = new_len % 64;
    memcpy(m_block, &shifted_message[block_nb << 6], rem_len);
    m_len = rem_len;
    m_tot_len += (block_nb + 1) << 6;
}

__device__
void SHA256::final(unsigned char *digest, char* shared_mem) {
    unsigned int block_nb;
    unsigned int pm_len;
    unsigned int len_b;
    char* m_block = M_BLOCK;
    int i;
    block_nb = (1 + ((64 - 9)
                     < (m_len % 64)));
    len_b = (m_tot_len + m_len) << 3;
    pm_len = block_nb << 6;
    memset(m_block + m_len, 0, pm_len - m_len);
    m_block[m_len] = 0x80;
    SHA2_UNPACK32(len_b, m_block + pm_len - 4);
    transform((unsigned char*)m_block, block_nb, shared_mem);
    for (i = 0 ; i < 8; i++) {
        SHA2_UNPACK32(M_H[i], &digest[i << 2]);
    }
}

__device__
void sha256(const char* input, int length, char* output, char* shared_mem)
{
    SHA256 ctx;
    ctx.init(shared_mem);
    ctx.update( (unsigned char*)input, length, shared_mem);
    ctx.final((unsigned char*)output, shared_mem);
}

extern "C" {
__global__ 
void Gpu_hash(const char* input, const unsigned int* sha_const, int length, int nonce_offset, int difficulty, int* result)
{
    __shared__ char shared_mem[49152];

    if (threadIdx.x == 0)
        for (int i = 0; i < 64; ++i)
            ((unsigned int*)shared_mem)[i] = sha_const[i];
    
    __syncthreads();

    const int nonce = nonce_offset + blockIdx.x*blockDim.x+threadIdx.x;
    
    unsigned char* nonce_input = NONCE_INPUT;
    for (int i = 0; i < length; ++i)
        nonce_input[i] = input[i];
    memcpy(nonce_input + length, (void*)&nonce, 4);

    unsigned char* digest = (unsigned char*)DIGEST;
 
    SHA256 ctx = SHA256();
    ctx.init(shared_mem);
    ctx.update(nonce_input, length + 4, shared_mem);
    ctx.final(digest, shared_mem);
    

    ctx.init(shared_mem);
    ctx.update(digest, 32, shared_mem);
    ctx.final(nonce_input, shared_mem);
    

    for (int i = 0; i < (difficulty >> 3); ++i)
        if (nonce_input[31 - i] != 0) return;
    if (nonce_input[31 - (difficulty >> 3)] <= 255 >> (difficulty & 7)) {
        *result = nonce;
    }
}
}




/*
 * Updated to C++, zedwood.com 2012
 * Based on Olivier Gay's version
 * See Modified BSD License below: 
 *
 * FIPS 180-2 SHA-224/256/384/512 implementation
 * Issue date:  04/30/2005
 * http://www.ouah.org/ogay/sha2/
 *
 * Copyright (C) 2005, 2007 Olivier Gay <olivier.gay@a3.epfl.ch>
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. Neither the name of the project nor the names of its contributors
 *    may be used to endorse or promote products derived from this software
 *    without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE PROJECT AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE PROJECT OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 */


